#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ int binarySearch(int *arr, int l, int r, int x) {
    if (r >= l) {
        int mid = (l + r )/2;

        if (arr[mid] == x)
            return mid;

        if (arr[mid] > x)
            return binarySearch(arr, l, mid-1, x);

        return binarySearch(arr, mid+1, r, x);
    }

    return -1;
}

__global__ void parallelExponentialSearch(int *arr, int size, int target, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int localRes = -1;
    if(tid < size) {
        int lowerBound = 1;
        while (lowerBound < target && arr[lowerBound] <= target) {
            lowerBound = lowerBound*2;
        }

        int upperBound = lowerBound < size-1 ? lowerBound : size-1;
        //printf("\nMin Val : %d\n", minVal);
        localRes =  binarySearch(arr, lowerBound/2, upperBound, target);
        *result = localRes;
    }
}

int main(int argc, char const *argv[]) {
    int *d_array, *d_result;
    int target = 118;
    int result;


    FILE *file = fopen("random_numbers.txt", "r");
    if (file == NULL) {
        printf("Failed to open the file for reading.\n");
        return 1;
    }

    int A[10000];
    int num_elements_A = 0;
    int val;

    while (fscanf(file, "%d", &val) != EOF) {
        A[num_elements_A] = val;
        num_elements_A++;
    }

    hipMalloc((void**)&d_array, num_elements_A * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_array, A, num_elements_A * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (num_elements_A + blockSize - 1) / blockSize;
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    parallelExponentialSearch<<<gridSize, blockSize>>>(d_array, num_elements_A, target, d_result);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);

    if (result != -1) {
        printf("Element %d found at postion : %d\n", target, result);
    } else {
        printf("Element %d not found in the array\n", target);
    }

    float eTMs;
    hipEventElapsedTime(&eTMs, start, stop);
    printf("\nTime Taken by the Kernal is : %f\n", eTMs);

    hipFree(d_array);
    hipFree(d_result);
    fclose(file);
    return 0;
}
